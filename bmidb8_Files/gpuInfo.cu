// get_compute_capability.cu
#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        std::cerr << "CUDA error getting device count: "
                  << hipGetErrorString(err) << "\n";
        return 1;
    }
    if (deviceCount == 0) {
        std::cout << "No CUDA devices found.\n";
        return 0;
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, dev);
        std::cout << "Device " << dev << ": " << prop.name
                  << "\n  Compute capability: "
                  << prop.major << "." << prop.minor << "\n";
    }
    return 0;
}
