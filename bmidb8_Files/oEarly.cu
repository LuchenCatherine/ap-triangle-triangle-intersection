// GPU_TriangleIntersection_NoEarlyExit.cu

#include <iostream>
#include <vector>
#include <array>
#include <fstream>
#include <sstream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

// Type aliases
using Vec3     = array<double,3>;
using Triangle = array<Vec3,3>;

// Tolerances
constexpr double EPSILON   = 1e-8;
constexpr double TOLERANCE = 1e-6;

// ---------------- Device helper functions ----------------

__device__ Vec3 crossD(const Vec3 &a, const Vec3 &b) {
    return {
        a[1]*b[2] - a[2]*b[1],
        a[2]*b[0] - a[0]*b[2],
        a[0]*b[1] - a[1]*b[0]
    };
}

__device__ double dotD(const Vec3 &a, const Vec3 &b) {
    return a[0]*b[0] + a[1]*b[1] + a[2]*b[2];
}

__device__ Vec3 subD(const Vec3 &a, const Vec3 &b) {
    return { a[0]-b[0], a[1]-b[1], a[2]-b[2] };
}

__device__ bool rayIntersectsTriangleD(
    const Vec3 &orig,
    const Vec3 &dir,
    const Triangle &tri,
    double &t, double &u, double &v
) {
    Vec3 edge1 = subD(tri[1], tri[0]);
    Vec3 edge2 = subD(tri[2], tri[0]);
    Vec3 pvec  = crossD(dir, edge2);
    double det = dotD(edge1, pvec);

    if (fabs(det) < EPSILON) return false;  // parallel

    double invDet = 1.0 / det;
    Vec3  tvec   = subD(orig, tri[0]);

    u = dotD(tvec, pvec) * invDet;
    if (u < 0.0 || u > 1.0) return false;

    Vec3 qvec = crossD(tvec, edge1);
    v = dotD(dir, qvec) * invDet;
    if (v < 0.0 || u + v > 1.0) return false;

    t = dotD(edge2, qvec) * invDet;
    return true;
}

__device__ bool trianglesIntersectD(
    const Triangle &A,
    const Triangle &B
) {
    // AABB overlap test
    double minAx = fmin(fmin(A[0][0], A[1][0]), A[2][0]);
    double minAy = fmin(fmin(A[0][1], A[1][1]), A[2][1]);
    double minAz = fmin(fmin(A[0][2], A[1][2]), A[2][2]);
    double maxAx = fmax(fmax(A[0][0], A[1][0]), A[2][0]);
    double maxAy = fmax(fmax(A[0][1], A[1][1]), A[2][1]);
    double maxAz = fmax(fmax(A[0][2], A[1][2]), A[2][2]);

    double minBx = fmin(fmin(B[0][0], B[1][0]), B[2][0]);
    double minBy = fmin(fmin(B[0][1], B[1][1]), B[2][1]);
    double minBz = fmin(fmin(B[0][2], B[1][2]), B[2][2]);
    double maxBx = fmax(fmax(B[0][0], B[1][0]), B[2][0]);
    double maxBy = fmax(fmax(B[0][1], B[1][1]), B[2][1]);
    double maxBz = fmax(fmax(B[0][2], B[1][2]), B[2][2]);

    if (maxAx < minBx - TOLERANCE || minAx > maxBx + TOLERANCE ||
        maxAy < minBy - TOLERANCE || minAy > maxBy + TOLERANCE ||
        maxAz < minBz - TOLERANCE || minAz > maxBz + TOLERANCE) {
        return false;
    }

    // Edge-vs-triangle tests (6 total)
    double t,u,v;
    for(int i=0;i<3;++i){
        Vec3 o = A[i];
        Vec3 d = subD(A[(i+1)%3], o);
        if(rayIntersectsTriangleD(o,d,B,t,u,v) && t>=0.0 && t<=1.0+TOLERANCE)
            return true;
    }
    for(int i=0;i<3;++i){
        Vec3 o = B[i];
        Vec3 d = subD(B[(i+1)%3], o);
        if(rayIntersectsTriangleD(o,d,A,t,u,v) && t>=0.0 && t<=1.0+TOLERANCE)
            return true;
    }
    return false;
}

// ---------------- Kernel WITHOUT early-exit flag checks ----------------

__global__ void triangleIntersectKernelNoEarlyExit(
    const Triangle *tri1, int n1,
    const Triangle *tri2, int n2,
    int *d_flag
) {
    int idx    = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // **No atomicAdd(d_flag,0)==0 check here**—
    // every thread will process its full range of i,j.
    for(int i=idx; i<n1; i+=stride){
        for(int j=0; j<n2; ++j){
            if(trianglesIntersectD(tri1[i], tri2[j])){
                // still use atomic to set the flag, but no bailout
                atomicExch(d_flag,1);
            }
        }
    }
}

// ---------------- Host utilities ----------------

vector<Triangle> readOFF(const string &file){
    ifstream in{file};
    string hdr; in>>hdr;
    if(hdr!="OFF") throw runtime_error("Not OFF");
    int verts,faces,edges;
    in>>verts>>faces>>edges;
    vector<Vec3> V(verts);
    for(int i=0;i<verts;++i)
        in>>V[i][0]>>V[i][1]>>V[i][2];
    vector<Triangle> out;
    for(int i=0;i<faces;++i){
        int cnt,a,b,c; in>>cnt>>a>>b>>c;
        if(cnt==3) out.push_back({V[a],V[b],V[c]});
    }
    return out;
}

int main(){
    // Load meshes
    auto A = readOFF("VH_F_cartilage_of_tertiary_bronchus_L.off");
    auto B = readOFF("VH_F_cartilage_of_tertiary_bronchus_R.off");

    // Device buffers
    Triangle *d_A, *d_B;
    int *d_flag;
    int   h_flag = 0;

    hipMalloc(&d_A,    A.size()*sizeof(Triangle));
    hipMalloc(&d_B,    B.size()*sizeof(Triangle));
    hipMalloc(&d_flag, sizeof(int));

    hipMemcpy(d_A,    A.data(), A.size()*sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_B,    B.data(), B.size()*sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_flag, &h_flag,    sizeof(int),                 hipMemcpyHostToDevice);

    // Launch parameters
    int blockSize = 256;
    int numBlocks = (A.size() + blockSize - 1) / blockSize;

    // Time only the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    triangleIntersectKernelNoEarlyExit<<<numBlocks, blockSize>>>(
        d_A, A.size(), d_B, B.size(), d_flag
    );

    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    cout << "Kernel time (no early-exit): " << ms << " ms\n";

    // Read back result
    hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);

    cout << (h_flag ? "Intersect\n" : "No intersect\n");

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_flag);

    return 0;
}
